#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_vector_types.h>
#include "ErrorCheck.h"

__constant__ u_int32_t c_morton256_x[256];
__constant__ u_int32_t c_morton256_y[256];
__constant__ u_int32_t c_morton256_z[256];

typedef union {
  uint2 u2;
  long long unsigned int l;
} _uint64;

extern "C"
void cudaConstants(const uint *x, const uint *y, const uint *z)
{
    ErrorCheck ec;
    ec.chk("setupConst start");
    hipMemcpyToSymbol(HIP_SYMBOL(c_morton256_x), x, sizeof(u_int32_t)*256,0); ec.chk("setupConst: c_morton256_x");
    hipMemcpyToSymbol(HIP_SYMBOL(c_morton256_y), y, sizeof(u_int32_t)*256,0); ec.chk("setupConst: c_morton256_y");
    hipMemcpyToSymbol(HIP_SYMBOL(c_morton256_z), z, sizeof(u_int32_t)*256,0); ec.chk("setupConst: c_morton256_z");

}

typedef u_int64_t uint64;
#define EMPTY_VOXEL 0
#define FULL_VOXEL 1
#define WORKING_VOXEL 2


/************************************************************
 * From Bonsai treecode
 * ********************************************************/
static __host__ __device__ uint2 dilate3(int value) {
  unsigned int x;
  uint2 key;

  // dilate first 10 bits

  x = value & 0x03FF;
  x = ((x << 16) + x) & 0xFF0000FF;
  x = ((x <<  8) + x) & 0x0F00F00F;
  x = ((x <<  4) + x) & 0xC30C30C3;
  x = ((x <<  2) + x) & 0x49249249;
  key.x = x;

  // dilate second 10 bits

  x = (value >> 10) & 0x03FF;
  x = ((x << 16) + x) & 0xFF0000FF;
  x = ((x <<  8) + x) & 0x0F00F00F;
  x = ((x <<  4) + x) & 0xC30C30C3;
  x = ((x <<  2) + x) & 0x49249249;
  key.y = x;

  return key;
}

//#if 0
//Morton order
static __host__  __device__ _uint64 get_key_morton(int4 crd) {
  _uint64 key, key1;
  key.u2  = dilate3(crd.x);

  key1.u2 = dilate3(crd.y);
  key.u2.x = key.u2.x | (key1.u2.x << 1);
  key.u2.y = key.u2.y | (key1.u2.y << 1);

  key1.u2 = dilate3(crd.z);
  key.u2.x = key.u2.x | (key1.u2.x << 2);
  key.u2.y = key.u2.y | (key1.u2.y << 2);

  return key;
}



template <typename T>
struct CAABox {
    T min;
    T max;
};

template<typename T>
__device__
CAABox<T> make_CAABox(const T &min, const T &max){CAABox<T> tmp; tmp.min = min; tmp.max = max; return tmp;}

template <typename T>
__device__ T d_clampval(const T& value, const T& low, const T& high) {
  return value < low ? low : (value > high ? high : value);
}

__device__
CAABox<float3> cudaComputeBoundingBox(const float3 &v0, const float3 &v1, const float3 &v2){
        CAABox<float3> answer;
        answer.min.x = fminf(v0.x,fminf(v1.x,v2.x));
        answer.min.y = fminf(v0.y,fminf(v1.y,v2.y));
        answer.min.z = fminf(v0.z,fminf(v1.z,v2.z));
        answer.max.x = fmaxf(v0.x,fmaxf(v1.x,v2.x));
        answer.max.y = fmaxf(v0.y,fmaxf(v1.y,v2.y));
        answer.max.z = fmaxf(v0.z,fmaxf(v1.z,v2.z));
        return answer;
}


template<bool COUNT_ONLY>
__device__
void voxelize_triangle(float3 v0, float3 v1, float3 v2,const uint64 morton_start, const uint64 morton_end, const float unitlength, int* voxels, uint64 *data, float sparseness_limit, uint *nfilled,
                       const uint3 &p_bbox_grid_min, const uint3 &p_bbox_grid_max, const float unit_div, const float3 &delta_p,	size_t data_max_items)

{


    // read triangle



//    if (use_data){
//        if (data.size() > data_max_items){
//            if (verbose){
//                cout << "Sparseness optimization side-array overflowed, reverting to slower voxelization." << endl;
//                cout << data.size() << " > " << data_max_items << endl;
//            }
//            use_data = false;
//        }
//    }


    // compute triangle bbox in world and grid
    const CAABox<float3> t_bbox_world = cudaComputeBoundingBox(v0, v1, v2);
    const int3 grid_min = make_int3((int)(t_bbox_world.min.x * unit_div),(int)(t_bbox_world.min.y * unit_div),(int)(t_bbox_world.min.z * unit_div));
    const int3 grid_max = make_int3((int)(t_bbox_world.max.x * unit_div),(int)(t_bbox_world.max.y * unit_div),(int)(t_bbox_world.max.z * unit_div));
    // clamp2
    const int3 clamp_grid_min = make_int3(d_clampval<int>(grid_min.x, p_bbox_grid_min.x, p_bbox_grid_max.x),
            d_clampval<int>(grid_min.y, p_bbox_grid_min.y, p_bbox_grid_max.y),
            d_clampval<int>(grid_min.z, p_bbox_grid_min.z, p_bbox_grid_max.z));
    const int3 clamp_grid_max = make_int3(d_clampval<int>(grid_max.x, p_bbox_grid_min.x, p_bbox_grid_max.x),
            d_clampval<int>(grid_max.y, p_bbox_grid_min.y, p_bbox_grid_max.y),
            d_clampval<int>(grid_max.z, p_bbox_grid_min.z, p_bbox_grid_max.z));
    const CAABox<int3> t_bbox_grid = make_CAABox<int3>(clamp_grid_min, clamp_grid_max);


    // COMMON PROPERTIES FOR THE TRIANGLE
    const float3 e0 = v1 - v0;
    const float3 e1 = v2 - v1;
    const float3 e2 = v0 - v2;
    float3 to_normalize = cross(e0,e1);
    const float3  n = normalize(to_normalize); // triangle normal
    // PLANE TEST PROPERTIES
    const float3 c = make_float3(n.x > 0 ? unitlength : 0.0f,
                        n.y > 0 ? unitlength : 0.0f,
                        n.z > 0 ? unitlength : 0.0f); // critical point
    const float d1 = dot(n , c - v0);
    const float d2 = dot(n, ((delta_p - c) - v0));
    // PROJECTION TEST PROPERTIES
    // XY plane
    const float2 n_xy_e0 = n.z < 0.0f ? -1.0f * make_float2(-1.0f*e0.y, e0.x) : make_float2(-1.0f*e0.y, e0.x);
    const float2 n_xy_e1 = n.z < 0.0f ? -1.0f * make_float2(-1.0f*e1.y, e1.x) : make_float2(-1.0f*e1.y, e1.x);
    const float2 n_xy_e2 = n.z < 0.0f ? -1.0f * make_float2(-1.0f*e2.y, e2.x) : make_float2(-1.0f*e2.y, e2.x);

    const float d_xy_e0 = (-1.0f * dot(n_xy_e0, make_float2(v0.x, v0.y))) + max(0.0f, unitlength*n_xy_e0.x) + max(0.0f, unitlength*n_xy_e0.y);
    const float d_xy_e1 = (-1.0f * dot(n_xy_e1, make_float2(v1.x, v1.y))) + max(0.0f, unitlength*n_xy_e1.x) + max(0.0f, unitlength*n_xy_e1.y);
    const float d_xy_e2 = (-1.0f * dot(n_xy_e2, make_float2(v2.x, v2.y))) + max(0.0f, unitlength*n_xy_e2.x) + max(0.0f, unitlength*n_xy_e2.y);
    // YZ plane
    const float2 n_yz_e0 = n.x < 0.0f ? -1.0f * make_float2(-1.0f*e0.z, e0.y) : make_float2(-1.0f*e0.z, e0.y);
    const float2 n_yz_e1 = n.x < 0.0f ? -1.0f * make_float2(-1.0f*e1.z, e1.y) : make_float2(-1.0f*e1.z, e1.y);
    const float2 n_yz_e2 = n.x < 0.0f ? -1.0f * make_float2(-1.0f*e2.z, e2.y) : make_float2(-1.0f*e2.z, e2.y);

    const float d_yz_e0 = (-1.0f * dot(n_yz_e0, make_float2(v0.y, v0.z))) + max(0.0f, unitlength*n_yz_e0.x) + max(0.0f, unitlength*n_yz_e0.y);
    const float d_yz_e1 = (-1.0f * dot(n_yz_e1, make_float2(v1.y, v1.z))) + max(0.0f, unitlength*n_yz_e1.x) + max(0.0f, unitlength*n_yz_e1.y);
    const float d_yz_e2 = (-1.0f * dot(n_yz_e2, make_float2(v2.y, v2.z))) + max(0.0f, unitlength*n_yz_e2.x) + max(0.0f, unitlength*n_yz_e2.y);
    // ZX plane
    const float2 n_zx_e0 = n.y < 0.0f ? -1.0f * make_float2(-1.0f*e0.x, e0.z) : make_float2(-1.0f*e0.x, e0.z);
    const float2 n_zx_e1 = n.y < 0.0f ? -1.0f * make_float2(-1.0f*e1.x, e1.z) : make_float2(-1.0f*e1.x, e1.z);
    const float2 n_zx_e2 = n.y < 0.0f ? -1.0f * make_float2(-1.0f*e2.x, e2.z) : make_float2(-1.0f*e2.x, e2.z);

    const float d_xz_e0 = (-1.0f * dot(n_zx_e0, make_float2(v0.z, v0.x))) + max(0.0f, unitlength*n_zx_e0.x) + max(0.0f, unitlength*n_zx_e0.y);
    const float d_xz_e1 = (-1.0f * dot(n_zx_e1, make_float2(v1.z, v1.x))) + max(0.0f, unitlength*n_zx_e1.x) + max(0.0f, unitlength*n_zx_e1.y);
    const float d_xz_e2 = (-1.0f * dot(n_zx_e2, make_float2(v2.z, v2.x))) + max(0.0f, unitlength*n_zx_e2.x) + max(0.0f, unitlength*n_zx_e2.y);

    // test possible grid boxes for overlap
    const int3 bbox_size = make_int3((t_bbox_grid.max.x - t_bbox_grid.min.x + 1), (t_bbox_grid.max.y - t_bbox_grid.min.y + 1), (t_bbox_grid.max.z - t_bbox_grid.min.z + 1));

    const int idx_cnt =  bbox_size.x * bbox_size.y * bbox_size.z;

#if 0
    const int z = t_bbox_grid.min.z;
    const int y = t_bbox_grid.min.y;
    const int x = t_bbox_grid.min.x;
    const _uint64 index = get_key_morton(make_int4(x,y,z,0));//cuda_mortonEncode_for(x,y,z);
    int idx = atomicInc(&nfilled[0], 100000000);//nfilled++;
    data[idx] = index.l;
#else

    for (int i=0; i<idx_cnt; i++){
        const int z = t_bbox_grid.min.z + i / (bbox_size.y * bbox_size.x);
        const int rem = i % (bbox_size.y * bbox_size.x);
        const int y = t_bbox_grid.min.y + (rem / bbox_size.x);
        const int x = t_bbox_grid.min.x + (rem % bbox_size.x);

        //const uint64 index = mortonEncode_magicbits(z, y, x);
        const _uint64 index = get_key_morton(make_int4(x,y,z,0));//cuda_mortonEncode_for(x,y,z);
        // TRIANGLE PLANE THROUGH BOX TEST
        const float3  p = make_float3(x*unitlength, y*unitlength, z*unitlength);
        const float nDOTp = dot(n , p);

        // PROJECTION TESTS
        // XY
        const float2 p_xy = make_float2(p.x, p.y);
        // YZ
        const float2 p_yz = make_float2(p.y, p.z);
        // XZ
        const float2 p_zx = make_float2(p.z, p.x);

        if (!(((nDOTp + d1) * (nDOTp + d2) > 0.0f)
                || ((dot(n_xy_e0 , p_xy) + d_xy_e0) < 0.0f)
                || ((dot(n_xy_e1 , p_xy) + d_xy_e1) < 0.0f)
                || ((dot(n_xy_e2 , p_xy) + d_xy_e2) < 0.0f)
                || ((dot(n_yz_e0 , p_yz) + d_yz_e0) < 0.0f)
                || ((dot(n_yz_e1 , p_yz) + d_yz_e1) < 0.0f)
                || ((dot(n_yz_e2 , p_yz) + d_yz_e2) < 0.0f)
                || ((dot(n_zx_e0 , p_zx) + d_xz_e0) < 0.0f)
                || ((dot(n_zx_e1 , p_zx) + d_xz_e1) < 0.0f)
                || ((dot(n_zx_e2 , p_zx) + d_xz_e2) < 0.0f)
                )){
            if (COUNT_ONLY == false){
                if (atomicCAS(&voxels[index.l - morton_start], EMPTY_VOXEL,FULL_VOXEL) == EMPTY_VOXEL){

                    //if (use_data){
                        int idx = atomicInc(&nfilled[0], 1000000000);//nfilled++;
                         data[idx] = index.l;
                    //}

                }
            }
        }

        __syncthreads();
    }
#endif
}

template<bool COUNT_ONLY>
__global__
void voxelize(const float3 *v0, const float3 *v1, const float3 *v2,const uint64 morton_start, const uint64 morton_end, const float unitlength, int* voxels, uint64 *data, float sparseness_limit, uint *nfilled,
               const uint3 p_bbox_grid_min, const uint3 p_bbox_grid_max, const float unit_div, const float3 delta_p,	size_t data_max_items, size_t num_triangles)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < num_triangles){
        voxelize_triangle<COUNT_ONLY>(v0[idx],v1[idx], v2[idx], morton_start, morton_end, unitlength, voxels, data, sparseness_limit, nfilled,
                          p_bbox_grid_min, p_bbox_grid_max, unit_div, delta_p, data_max_items);
    }
//    const int z = idx / (128*128);
//    const int rem = idx % (128*128);
//    const int y = (rem / 128);
//    const int x = (rem % 128);
//    data[idx] = cuda_mortonEncode_for(x,y,z);
}


#include <tbb/atomic.h>
#include <tbb/tbb.h>
extern "C"
void cudaRun(const float3* d_v0, const float3*d_v1, const float3*d_v2,const uint64 morton_start, const uint64 morton_end, const float unitlength, tbb::atomic<char> *voxels, tbb::concurrent_vector<uint64> &data, float sparseness_limit, bool &use_data, tbb::atomic<size_t> &nfilled,
             const uint3 &p_bbox_grid_min, const uint3 &p_bbox_grid_max, const float unit_div, const float3 &delta_p,	size_t data_max_items, size_t num_triangles)
{
    ErrorCheck ec;
    ec.chk("cudaRun");
    int *d_voxels, *h_voxels;
    hipMalloc( (void**) &d_voxels, sizeof(int)*(morton_end - morton_start));    ec.chk("voxel malloc");

    h_voxels = new int[morton_end - morton_start];

    uint64 *d_data;
    hipMalloc( (void**) &d_data, sizeof(uint64)*data.size());     ec.chk("data malloc");

    uint64 *h_data = new uint64[data.size()];

    uint *d_nfilled;
    hipMalloc( (void**) &d_nfilled, sizeof(uint));     ec.chk("nfilled malloc");

    hipMemset(d_nfilled,0, sizeof(uint));
    hipMemset(d_voxels, EMPTY_VOXEL, sizeof(int)*(morton_end - morton_start));
    hipMemset(d_data, 0, sizeof(uint64)*data.size());
    ec.chk("memory finished");
    //get count
    voxelize<false><<<10000,32>>>(d_v0, d_v1, d_v2, morton_start, morton_end, unitlength, d_voxels, d_data, use_data, d_nfilled,
                    p_bbox_grid_min, p_bbox_grid_max, unit_div, delta_p, data_max_items, num_triangles);

    hipDeviceSynchronize();
    ec.chk("voxelize done" );
    uint h_nfilled = 0;
    hipMemcpy(&h_nfilled, d_nfilled, sizeof(uint), hipMemcpyDeviceToHost);
    //nfilled = h_nfilled;
//    hipMemcpy(h_voxels, d_voxels, sizeof(int) * (morton_end - morton_start), hipMemcpyDeviceToHost);
//    for (int i=0; i<(morton_end - morton_start); i++){
//        voxels[i] = (char)h_voxels[i];
//    }

    hipMemcpy(h_data, d_data, data.size()*sizeof(uint64), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
//    std::sort(data.begin(), data.end());
//    std::sort(h_data, h_data + data.size());
    for (int i=0; i<data.size(); i++){
        data[i] = h_data[i];
    }


    hipFree(d_voxels);
    hipFree(d_data);
    hipFree(d_nfilled);
    delete h_voxels;
    delete h_data;

}
